#include "hip/hip_runtime.h"
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include <benchmark/benchmark.h>


__global__ void gpu_addition(int *a, int *b, int *c){
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void cpu_addition(int *a, int *b, int *c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

static void BM_cpu_addition(benchmark::State& state) {
    int n = 10'000;

    int a[n];
    int b[n];
    int c[n];
    std::fill_n(a, n, 1);
    std::fill_n(b, n, 2);
    std::fill_n(c, n, 0);
    
    for (auto _ : state) {
        cpu_addition(a, b, c, n);
    }
}

static void BM_gpu_addition(benchmark::State& state) {
    int n = 10'000;

    // allocate memory on cpu
    int a[n];
    int b[n];
    int c[n];
    std::fill_n(a, n, 1);
    std::fill_n(b, n, 2);
    std::fill_n(c, n, 0);

    // allocate memory on gpu
    int *cuda_a = 0;
    int *cuda_b = 0;
    int *cuda_c = 0;
    assert(hipMalloc(&cuda_a, sizeof(a)) == hipSuccess);
    assert(hipMalloc(&cuda_b, sizeof(b)) == hipSuccess);
    assert(hipMalloc(&cuda_c, sizeof(c)) == hipSuccess);
    assert(hipMemcpy(cuda_a, a, sizeof(a), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(cuda_b, b, sizeof(b), hipMemcpyHostToDevice) == hipSuccess);
    assert(hipMemcpy(cuda_c, c, sizeof(c), hipMemcpyHostToDevice) == hipSuccess);
    
    // set execution configuration
    int grid_size = 1;  // how many threads per block
    int block_size = n;  // how many blocks

    // TODO: interestingly, it seems like allocating a small block size < 10'000
    // causes the function to run *much* slower. I'm not sure why, will have to
    // do more investigation

    for (auto _ : state) {
        // call function
        gpu_addition <<< grid_size, block_size >>> (cuda_a, cuda_b, cuda_c);
        hipDeviceSynchronize();
    }

    // copy back to host
    assert(hipMemcpy(c, cuda_c, sizeof(c), hipMemcpyDeviceToHost) == hipSuccess);
}


/*
int main() {
    int a[] = {1, 2, 3};
    int b[] = {5, 6, 7};
    int c[3] = { 0 };

    int *cuda_a = 0;
    int *cuda_b = 0;
    int *cuda_c = 0;

    return 0;
}
*/

BENCHMARK(BM_cpu_addition);
BENCHMARK(BM_gpu_addition);
BENCHMARK_MAIN();