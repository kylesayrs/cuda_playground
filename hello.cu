#include <stdio.h>

#include <hip/hip_runtime.h>


__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    printf("Hello World from CPU!\n");

    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}